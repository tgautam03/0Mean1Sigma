void CUDA_CHECK(hipError_t err) 
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}

// Device array pointers
float* d_A;
float* d_B;
float* d_C;

// Device memory allocation
hipError_t err_A = hipMalloc((void**) &d_A, N*N*sizeof(float));
CUDA_CHECK(err_A);

hipError_t err_B = hipMalloc((void**) &d_B, N*N*sizeof(float));
CUDA_CHECK(err_B);

hipError_t err_C = hipMalloc((void**) &d_C, N*N*sizeof(float));
CUDA_CHECK(err_C);