#include "hip/hip_runtime.h"
__global__ void tiled_mat_mul_kernel(float* A, float* B, float* C, int N1, int N2, int N3)
{
    // Details regarding this thread
    int by = blockIdx.y;
    int bx = blockIdx.x; 

    int ty = threadIdx.y;
    int tx = threadIdx.x; 

    // Working on C[i,j]
    int i = TILE_WIDTH*by + ty;
    int j = TILE_WIDTH*bx + tx;

    // Allocating shared memory
    __shared__ float sh_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sh_B[TILE_WIDTH][TILE_WIDTH];

    // Parallel mat mul
    float value = 0;
    for (int phase = 0; phase < ceil((float)N2/TILE_WIDTH); phase++)
    {
        // Load Tiles into shared memory
        if ((i < N1) && ((phase*TILE_WIDTH+tx) < N2))
          sh_A[ty][tx] = A[(i)*N2 + phase*TILE_WIDTH+tx];
        else
          sh_A[ty][tx] = 0.0f;

        if (((phase*TILE_WIDTH + ty) < N2) && (j < N3))
          sh_B[ty][tx] = B[(phase*TILE_WIDTH + ty)*N3+j];
        else
          sh_B[ty][tx] = 0.0f;
        __syncthreads();

        // Dot product
        for (int k = 0; k < TILE_WIDTH; k++)
            value += sh_A[ty][k] * sh_B[k][tx];
        __syncthreads();
    }
    // Assigning calculated value
    if ((i < N1) && (j < N3))
      C[i*N3+j] = value;
}