// Copying A and B to device memory
hipError_t err_A_ = hipMemcpy(d_A, A, N*N*sizeof(float), hipMemcpyHostToDevice);
CUDA_CHECK(err_A_);

hipError_t err_B_ = hipMemcpy(d_B, B, N*N*sizeof(float), hipMemcpyHostToDevice);
CUDA_CHECK(err_B_);

// Kernel execution

// Copy back results
hipError_t err_C_ = hipMemcpy(C, d_C, N*N*sizeof(float), hipMemcpyDeviceToHost);
CUDA_CHECK(err_C_);