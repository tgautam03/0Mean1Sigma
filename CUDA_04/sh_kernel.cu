#include "hip/hip_runtime.h"
__global__ void tiled_sq_mat_mul_kernel(float* A, float* B, float* C, int N)
{
    // Details regarding this thread
    int by = blockIdx.y;
    int bx = blockIdx.x; 

    int ty = threadIdx.y;
    int tx = threadIdx.x; 

    // Working on C[i,j]
    int i = blockDim.y*by + ty;
    int j = blockDim.x*bx + tx;

    // Allocating shared memory
    __shared__ float sh_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sh_B[TILE_WIDTH][TILE_WIDTH];

    // Parallel mat mul
    float value = 0;
    // Splitting data into smaller tiles
    for (int phase = 0; phase < N/TILE_WIDTH; phase++)
    {
        // Load Tiles into shared memory
        sh_A[ty][tx] = A[(i)*N + phase*TILE_WIDTH+tx];
        sh_B[ty][tx] = B[(phase*TILE_WIDTH + ty)*N+j];
        __syncthreads();

        // Dot product with data in shared memory
        for (int k = 0; k < TILE_WIDTH; k++)
            value += sh_A[ty][k] * sh_B[k][tx];
        __syncthreads();
    }
    // Assigning calculated value
    C[i*N+j] = value;
}